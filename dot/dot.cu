#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <chrono>

using namespace std;

const int N = 1024;

__global__ void cudaDotKernel(const float* a, const float* b, float* c, const uint size){

	__shared__ float tmp[N];

	uint thread_index = blockIdx.x * blockDim.x + threadIdx.x;
	tmp[thread_index] = a[thread_index]*b[thread_index];
	__syncthreads();

	if(threadIdx.x == 0){
		int sum = 0;
		for(int i = 0; i < N; i++) sum += tmp[i];
		*c = sum;
		
	}

	thread_index += blockDim.x * gridDim.x;
}

void cudaCallDotKernel(
	const uint block_count,
	const uint per_block_thread_count,
	const float* a,
	const float* b,
	float* c,
	const uint size
){
	cudaDotKernel<<<block_count, per_block_thread_count>>>(a, b, c, size);
}

int main(int argc, char** argv){
	hipDeviceProp_t prop;
	int device;
	hipGetDevice(&device);
	hipGetDeviceProperties(&prop, device);

	const uint per_block_thread_count = prop.maxThreadsPerBlock;
	const uint max_block_count = 1024;

	int* a = (int*)malloc(N*sizeof(int));
	int* b = (int*)malloc(N*sizeof(int));
	int c;

	for(int i = 0; i < N; i++){
		a[i] = (int)i;
		b[i] = (int)i;
		c[i] = (int)i;
	}

	int* dev_a;
	int* dev_b;
	int dev_c;

	hipMalloc((void**) &dev_a, N*sizeof(int));
	hipMalloc((void**) &dev_b, N*sizeof(int));
	hipMalloc((void**) &dev_c, sizeof(int));

	hipMemcpy(dev_a, a, int*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, int*sizeof(int), hipMemcpyHostToDevice);

	uint block_count = min(max_block_count, (uint)ceil(N / (int)per_block_thread_count));

	auto start = std::chrono::high_resolution_clock::now();
	cudaCallDotKernel(
		block_count,
		per_block_thread_count,
		dev_a,
		dev_b,
		dev_c,
		N
	);
	auto end = std::chrono::high_resolution_clock::now();

	hipMemcpy(c, dev_c, sizeof(float), hipMemcpyDeviceToHost);
	
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    std::cout << "Time taken: " << duration.count() << " microseconds" << std::endl;

	free(a);
	free(b);
	free(c);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}
